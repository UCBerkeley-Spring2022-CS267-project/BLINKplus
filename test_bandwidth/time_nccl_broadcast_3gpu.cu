#include <stdio.h>
#include <cstdlib>
#include <vector>
#include <chrono>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


int main(int argc, char* argv[])
{
    if ( argc != 7 )
    {
        printf("Usage ./time_nccl_broadcast GU1 GPU2 GPU2 NUM_WARMUP NUM_ITER TOTAL_DATA_SIZE\n");
        exit(1);
    }

    setenv( "NCCL_PROTO", "Simple", 1);
    setenv( "NCCL_ALGO", "Tree", 1 );
    //setenv( "NCCL_DEBUG", "Info", 1);
    //setenv( "NCCL_DEBUG_SUBSYS", "ALL", 1);
    setenv( "NCCL_GRAPH_FILE", "graph_input.xml", 1 );
    setenv( "NCCL_GRAPH_DUMP_FILE", "graph_output.xml", 1 );

    printf("%s:: NCCL Version %d.%d.%d\n", __func__, NCCL_MAJOR, NCCL_MINOR, NCCL_PATCH );

    // User allocate resources
    int total_data_size = atoi( argv[6] )*1024*1024;
    int num_warmup = atoi( argv[4] );
    int num_iters = atoi( argv[5] );
  
    int user_group_num_comm = 3;
    std::vector<int> user_group_devs = { atoi(argv[1]), atoi( argv[2]), atoi( argv[3]) };
    std::vector<ncclComm_t> user_group_comms( user_group_num_comm );
    std::vector<hipStream_t> user_group_streams( user_group_num_comm );

    printf("%s:: User GPU %d, %d, %d\n", __func__, user_group_devs[0], user_group_devs[1], user_group_devs[2]);

    printf("%s:: Init stream data\n", __func__ );
    for ( int i = 0; i < user_group_num_comm; ++i )
    {
        CUDACHECK(hipSetDevice( user_group_devs[ i ] ));
        CUDACHECK(hipStreamCreate( &(user_group_streams[i]) ));
    }

    printf("=========%s:: Initial data of size %d MB=========\n", __func__, int(atoi(argv[6]) * sizeof(uint8_t)));
    // currently ignore none dividable data case
    int chunk_data_size = total_data_size;

    uint8_t** user_group_sendbuff = (uint8_t**)malloc(user_group_num_comm * sizeof(uint8_t*));
    uint8_t** user_group_recvbuff = (uint8_t**)malloc(user_group_num_comm * sizeof(uint8_t*));

    std::vector<uint8_t> h_sendbuff( chunk_data_size );
    for ( int i = 0; i < chunk_data_size; ++i )
    {
      h_sendbuff[ i ] = i;
    }

    for ( int comm_i = 0; comm_i < user_group_num_comm; ++comm_i )
    {
      CUDACHECK(hipSetDevice( user_group_devs[ comm_i ] ));
      CUDACHECK(hipMalloc( (user_group_sendbuff + comm_i), chunk_data_size * sizeof(uint8_t) ));
      CUDACHECK(hipMalloc( (user_group_recvbuff + comm_i), chunk_data_size * sizeof(uint8_t)));
      CUDACHECK(hipMemcpy( user_group_sendbuff[ comm_i ], h_sendbuff.data(), chunk_data_size * sizeof(uint8_t), hipMemcpyHostToDevice ));
      CUDACHECK(hipMemset( user_group_recvbuff[ comm_i ], 0, chunk_data_size * sizeof(uint8_t)));    
    }

    NCCLCHECK(ncclCommInitAll( user_group_comms.data(), user_group_num_comm, user_group_devs.data() ));


    printf("=====Start WarmUp Iters: %d =====\n", num_warmup);
    for (int iter = 0; iter < num_warmup; iter++) 
    {
      NCCLCHECK(ncclGroupStart());
      for ( int i = 0; i < user_group_num_comm; ++i ) 
      {
          NCCLCHECK(ncclBroadcast((const void*)user_group_sendbuff[ i ], \
                                  (void*)user_group_recvbuff[ i ], \
                                  chunk_data_size, ncclInt8, user_group_devs[ 0 ], \
                                  user_group_comms[i], \
                                  user_group_streams[i]));
      }
      NCCLCHECK(ncclGroupEnd());

    }

    printf("%s:: Synchronize warmup\n", __func__ );
    for ( int i = 0; i < user_group_num_comm; ++i ) 
    {
      CUDACHECK(hipSetDevice( user_group_devs[i]));
      CUDACHECK(hipStreamSynchronize( user_group_streams[i] ));
    }

    printf("=====End WarmUp=====\n");

    // Start timing
    printf("=====Start Timing, Iters: %d ======\n", num_iters);
    auto start = std::chrono::high_resolution_clock::now();
    for (int iter = 0; iter < num_iters; iter++) 
    {
        NCCLCHECK(ncclGroupStart());
        for ( int i = 0; i < user_group_num_comm; ++i )
        {
            NCCLCHECK(ncclBroadcast((const void*)user_group_sendbuff[ i ], \
                                    (void*)user_group_recvbuff[ i ], \
                                    chunk_data_size, ncclInt8, user_group_devs[ 0 ], \
                                    user_group_comms[i], \
                                    user_group_streams[i]));
        }
        NCCLCHECK(ncclGroupEnd());
    }

    for ( int i = 0; i < user_group_num_comm; ++i ) 
    {
      CUDACHECK(hipSetDevice( user_group_devs[i]));
      CUDACHECK(hipStreamSynchronize( user_group_streams[i]));
    }

    // End timing
    {
      printf("=====End Timing User======\n");
      auto delta = std::chrono::high_resolution_clock::now() - start;
      double deltaSec = std::chrono::duration_cast<std::chrono::duration<double>>(delta).count();
      deltaSec = deltaSec / num_iters;
      double timeUsec = deltaSec*1.0E6;
      double bw = total_data_size * sizeof(uint8_t) / 1.0E9 / deltaSec;
      printf("%s:: Average of %d Iters, data: %d MB,  Elapsed Time: %7.5f (us), BandWidth: %7.5f (GB/s)\n", \
                  __func__, num_iters, int(atoi(argv[6]) * sizeof(uint8_t)), timeUsec,  bw);  
    }

    printf("%s:: check data correctness after stream synchronize\n", __func__);
    std::vector<uint8_t> h_recvbuff( chunk_data_size );

    for ( int comm_i = 0; comm_i < user_group_num_comm; ++comm_i )
    {
      CUDACHECK( hipMemcpy( h_recvbuff.data(), user_group_recvbuff[ comm_i ], chunk_data_size * sizeof( uint8_t ), hipMemcpyDeviceToHost ));
      for ( int i = 0; i < h_recvbuff.size(); ++i )
      {
        if ( h_recvbuff[i] != h_sendbuff[i] )
        {
          printf("%s:: Check recv on user group comm %d failed, expected %d but have %d\n", __func__, comm_i, h_sendbuff[i], h_recvbuff[i] );
        }
      }
    }

    for ( int comm_i = 0; comm_i < user_group_num_comm; ++comm_i )
    {
      CUDACHECK(hipSetDevice( user_group_devs[ comm_i ] ));
      CUDACHECK(hipFree( user_group_sendbuff[ comm_i ] ));
      CUDACHECK(hipFree( user_group_recvbuff[ comm_i ] ));
    }

    for ( int i = 0; i < user_group_num_comm; ++i ) 
    {
        ncclCommDestroy( user_group_comms[i]);
    }

    printf("%s:: Success \n", __func__);
    return 0;
}