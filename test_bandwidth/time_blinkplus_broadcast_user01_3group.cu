#include <stdio.h>
#include <cstdlib>
#include <vector>
#include <chrono>
#include <string>
#include <stdexcept> // std::runtime_error
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


struct blinkplusHelperGroup
{
    std::vector<int> devs;
    std::vector<ncclComm_t> comms;
    std::vector<uint8_t*> sendbuff;
    std::vector<uint8_t*> recvbuff;
    std::vector<hipStream_t> streams;
    std::string graph_filepath;
    int num_comms;

    blinkplusHelperGroup( const char* graph_filepath_cstr, std::vector<int> devs )
    {
        if ( std::getenv(graph_filepath_cstr) == nullptr )
        {
            throw std::runtime_error( std::string(graph_filepath_cstr) + " unset\n");
        }
        this->graph_filepath = std::getenv(graph_filepath_cstr);
        this->devs = devs;
        this->comms.resize( this->devs.size() );
        this->sendbuff.resize( this->devs.size() );
        this->recvbuff.resize( this->devs.size() );
        this->streams.resize( this->devs.size() );
        this->num_comms = this->devs.size();
    }
};


std::vector<blinkplusHelperGroup> blinkplusHelperGroupsContainer;


ncclResult_t blinkplusStreamSynchronize( )
{
  for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
  {  
    for ( int comm_j = 0; comm_j < blinkplusHelperGroupsContainer.at( group_i ).num_comms; comm_j++ )
    {
      #ifndef NDEBUG
      printf("%s:: group %d, comm %d call stream sync on dev %d\n", \
        __func__, group_i, comm_j, blinkplusHelperGroupsContainer.at( group_i ).devs.at( comm_j ) );
      #endif

      CUDACHECK(hipSetDevice( blinkplusHelperGroupsContainer.at( group_i ).devs.at( comm_j ) ));
      CUDACHECK(hipStreamSynchronize( blinkplusHelperGroupsContainer.at( group_i ).streams.at( comm_j ) ));
    }
  }
  return ncclSuccess;
}


int main(int argc, char* argv[])
{
    if ( argc != 6 )
    {
        printf("Usage ./time_nccl_broadcast_2group 0 1 NUM_WARMUP NUM_ITER TOTAL_DATA_SIZE\n");
        exit(1);
    }

    setenv( "NCCL_PROTO", "Simple", 1);
    setenv( "NCCL_ALGO", "Tree", 1 );

    printf("%s:: NCCL Version %d.%d.%d\n", __func__, NCCL_MAJOR, NCCL_MINOR, NCCL_PATCH );

    // User allocate resources
    int total_data_size = atoi( argv[5] )*1024*1024;
    int num_warmup = atoi( argv[3] );
    int num_iters = atoi( argv[4] );
  
    blinkplusHelperGroupsContainer.clear();
    blinkplusHelperGroupsContainer.reserve(2);

    blinkplusHelperGroup user_group( "BLINKPLUS_GRAPH_FILE_CHAIN_01", std::vector<int>{0,1});
    blinkplusHelperGroupsContainer.emplace_back( "BLINKPLUS_GRAPH_FILE_CHAIN_021", std::vector<int>{0,2,1} );
    blinkplusHelperGroupsContainer.emplace_back( "BLINKPLUS_GRAPH_FILE_CHAIN_031", std::vector<int>{0,3,1} );

    printf("%s:: User GPU %d, %d\n", __func__, user_group.devs[0], user_group.devs[1]);
    printf("%s:: Helper 1 GPU %d, %d, %d\n", __func__, blinkplusHelperGroupsContainer[ 0 ].devs[0], \
                                                       blinkplusHelperGroupsContainer[ 0 ].devs[1], \
                                                       blinkplusHelperGroupsContainer[ 0 ].devs[2]);
    printf("%s:: Helper 2 GPU %d, %d, %d\n", __func__, blinkplusHelperGroupsContainer[ 1 ].devs[0], \
                                                       blinkplusHelperGroupsContainer[ 1 ].devs[1], \
                                                       blinkplusHelperGroupsContainer[ 1 ].devs[2]);

    printf("%s:: Init stream data\n", __func__ );
    for ( int i = 0; i < user_group.num_comms; ++i )
    {
        CUDACHECK(hipSetDevice( user_group.devs[ i ] ));
        CUDACHECK(hipStreamCreate( &(user_group.streams[i]) ));
    }

    for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
    {
        for ( int i = 0; i < blinkplusHelperGroupsContainer.at( group_i ).num_comms; ++i )
        {
            CUDACHECK(hipSetDevice( blinkplusHelperGroupsContainer.at( group_i ).devs[ i ] ));
            CUDACHECK(hipStreamCreate( &(blinkplusHelperGroupsContainer.at( group_i ).streams[i]) ));
        }
    }

    printf("=========%s:: Initial data of size %d MB=========\n", __func__, int(atoi(argv[5]) * sizeof(uint8_t)));
    // currently ignore none dividable data case
    int chunk_data_size = total_data_size / 3;

    std::vector<uint8_t> h_sendbuff( chunk_data_size );
    for ( int i = 0; i < chunk_data_size; ++i )
    {
      h_sendbuff[ i ] = i;
    }

    for ( int comm_i = 0; comm_i < user_group.num_comms; ++comm_i )
    {
      CUDACHECK(hipSetDevice( user_group.devs[ comm_i ] ));
      CUDACHECK(hipMalloc( (user_group.sendbuff.data() + comm_i), chunk_data_size * sizeof(uint8_t) ));
      CUDACHECK(hipMalloc( (user_group.recvbuff.data() + comm_i), chunk_data_size * sizeof(uint8_t)));
      CUDACHECK(hipMemcpy( user_group.sendbuff[ comm_i ], h_sendbuff.data(), chunk_data_size * sizeof(uint8_t), hipMemcpyHostToDevice ));
      CUDACHECK(hipMemset( user_group.recvbuff[ comm_i ], 0, chunk_data_size * sizeof(uint8_t)));    
    }

    // Allocate buffer for user group
    

    for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
    {
        for ( int comm_i = 0; comm_i < blinkplusHelperGroupsContainer.at( group_i ).num_comms; ++comm_i )
        {
        CUDACHECK(hipSetDevice( blinkplusHelperGroupsContainer.at( group_i ).devs[ comm_i ] ));
        CUDACHECK(hipMalloc( (blinkplusHelperGroupsContainer.at( group_i ).sendbuff.data() + comm_i), chunk_data_size * sizeof(uint8_t) ));
        CUDACHECK(hipMalloc( (blinkplusHelperGroupsContainer.at( group_i ).recvbuff.data() + comm_i), chunk_data_size * sizeof(uint8_t)));
        CUDACHECK(hipMemcpy( blinkplusHelperGroupsContainer.at( group_i ).sendbuff[ comm_i ], h_sendbuff.data(), chunk_data_size * sizeof(uint8_t), hipMemcpyHostToDevice ));
        CUDACHECK(hipMemset( blinkplusHelperGroupsContainer.at( group_i ).recvbuff[ comm_i ], 0, chunk_data_size * sizeof(uint8_t)));    
        }        
    }

    setenv( "NCCL_GRAPH_FILE", std::getenv("BLINKPLUS_GRAPH_FILE_CHAIN_01") , 1 );
    NCCLCHECK(ncclCommInitAll( user_group.comms.data(), user_group.num_comms, user_group.devs.data() ));

    for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
    {
        setenv( "NCCL_GRAPH_FILE", blinkplusHelperGroupsContainer.at( group_i ).graph_filepath.c_str() , 1 );
        NCCLCHECK(ncclCommInitAll( blinkplusHelperGroupsContainer.at( group_i ).comms.data(), \
                                   blinkplusHelperGroupsContainer.at( group_i ).num_comms, \
                                   blinkplusHelperGroupsContainer.at( group_i ).devs.data() ));
    }

    printf("=====Start WarmUp Iters: %d =====\n", num_warmup);
    for (int iter = 0; iter < num_warmup; iter++) 
    {
      NCCLCHECK(ncclGroupStart());
      for ( int i = 0; i < user_group.num_comms; ++i ) 
      {
          NCCLCHECK(ncclBroadcast((const void*)user_group.sendbuff[ i ], \
                                  (void*)user_group.recvbuff[ i ], \
                                  chunk_data_size, ncclInt8, user_group.devs[ 0 ], \
                                  user_group.comms[i], \
                                  user_group.streams[i]));
      }
      NCCLCHECK(ncclGroupEnd());

      for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
      {
        NCCLCHECK(ncclGroupStart());
        for ( int i = 0; i < blinkplusHelperGroupsContainer.at( group_i).num_comms; ++i ) 
        {
            NCCLCHECK(ncclBroadcast((const void*)blinkplusHelperGroupsContainer.at( group_i).sendbuff[ i ], \
                                    (void*)blinkplusHelperGroupsContainer.at( group_i).recvbuff[ i ], \
                                    chunk_data_size, ncclInt8, blinkplusHelperGroupsContainer.at( group_i).devs[ 0 ], \
                                    blinkplusHelperGroupsContainer.at( group_i).comms[i], \
                                    blinkplusHelperGroupsContainer.at( group_i).streams[i]));
        }
        NCCLCHECK(ncclGroupEnd());
      }
    }

    for ( int i = 0; i < user_group.num_comms; ++i ) 
    {
      CUDACHECK(hipSetDevice( user_group.devs[i]));
      CUDACHECK(hipStreamSynchronize( user_group.streams[i] ));
    }

    for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
    {
        for ( int i = 0; i < blinkplusHelperGroupsContainer.at( group_i ).num_comms; ++i ) 
        {
          CUDACHECK(hipSetDevice( blinkplusHelperGroupsContainer.at( group_i ).devs[i] ));
          CUDACHECK(hipStreamSynchronize( blinkplusHelperGroupsContainer.at( group_i ).streams[i] ));
        }    
    }

    printf("=====End WarmUp=====\n");

    // Start timing
    printf("=====Start Timing, Iters: %d ======\n", num_iters);
    auto start = std::chrono::high_resolution_clock::now();
    for (int iter = 0; iter < num_iters; iter++) 
    {
      NCCLCHECK(ncclGroupStart());
      for ( int i = 0; i < user_group.num_comms; ++i ) 
      {
          NCCLCHECK(ncclBroadcast((const void*)user_group.sendbuff[ i ], \
                                  (void*)user_group.recvbuff[ i ], \
                                  chunk_data_size, ncclInt8, user_group.devs[ 0 ], \
                                  user_group.comms[i], \
                                  user_group.streams[i]));
      }
      NCCLCHECK(ncclGroupEnd());

      for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
      {
        NCCLCHECK(ncclGroupStart());
        for ( int i = 0; i < blinkplusHelperGroupsContainer.at( group_i).num_comms; ++i ) 
        {
            NCCLCHECK(ncclBroadcast((const void*)blinkplusHelperGroupsContainer.at( group_i).sendbuff[ i ], \
                                    (void*)blinkplusHelperGroupsContainer.at( group_i).recvbuff[ i ], \
                                    chunk_data_size, ncclInt8, blinkplusHelperGroupsContainer.at( group_i).devs[ 0 ], \
                                    blinkplusHelperGroupsContainer.at( group_i).comms[i], \
                                    blinkplusHelperGroupsContainer.at( group_i).streams[i]));
        }
        NCCLCHECK(ncclGroupEnd());
      }
    }

    for ( int i = 0; i < user_group.num_comms; ++i ) 
    {
      CUDACHECK(hipSetDevice( user_group.devs[i]));
      CUDACHECK(hipStreamSynchronize( user_group.streams[i] ));
    }

    NCCLCHECK( blinkplusStreamSynchronize() );

    // End timing
    {
      printf("=====End Timing======\n");
      auto delta = std::chrono::high_resolution_clock::now() - start;
      double deltaSec = std::chrono::duration_cast<std::chrono::duration<double>>(delta).count();
      deltaSec = deltaSec / num_iters;
      double timeUsec = deltaSec*1.0E6;
      double bw = total_data_size * sizeof(uint8_t) / 1.0E9 / deltaSec;
      printf("%s:: Average of %d Iters, data: %d MB,  Elapsed Time: %7.5f (us), BandWidth: %7.5f (GB/s)\n", \
                  __func__, num_iters, int(atoi(argv[5]) * sizeof(uint8_t)), timeUsec,  bw);
    }

    printf("%s:: check data correctness after stream synchronize\n", __func__);
    std::vector<uint8_t> h_recvbuff( chunk_data_size );

    for ( int comm_i = 0; comm_i < user_group.num_comms; ++comm_i )
    {
      CUDACHECK( hipMemcpy( h_recvbuff.data(), user_group.recvbuff[ comm_i ], chunk_data_size * sizeof( uint8_t ), hipMemcpyDeviceToHost ));
      for ( int i = 0; i < h_recvbuff.size(); ++i )
      {
        if ( h_recvbuff[i] != h_sendbuff[i] )
        {
          printf("%s:: Check recv on user group comm %d failed, expected %d but have %d\n", __func__, comm_i, h_sendbuff[i], h_recvbuff[i] );
        }
      }
    }

    for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
    {
        for ( int comm_i = 0; comm_i < blinkplusHelperGroupsContainer.at( group_i ).num_comms; ++comm_i )
        {
            CUDACHECK( hipMemcpy( h_recvbuff.data(), blinkplusHelperGroupsContainer.at( group_i ).recvbuff[ comm_i ], chunk_data_size * sizeof( uint8_t ), hipMemcpyDeviceToHost ));
            for ( int i = 0; i < h_recvbuff.size(); ++i )
            {
                if ( h_recvbuff[i] != h_sendbuff[i] )
                {
                printf("%s:: Check recv on helper group comm %d failed, expected %d but have %d\n", __func__, comm_i, h_sendbuff[i], h_recvbuff[i] );
                }
            }
        }        
    }

    for ( int comm_i = 0; comm_i < user_group.num_comms; ++comm_i )
    {
      CUDACHECK(hipSetDevice( user_group.devs[ comm_i ] ));
      CUDACHECK(hipFree( user_group.sendbuff[ comm_i ] ));
      CUDACHECK(hipFree( user_group.recvbuff[ comm_i ] ));
    }

    for ( int comm_i = 0; comm_i < user_group.num_comms; ++comm_i ) 
    {
        NCCLCHECK( ncclCommDestroy( user_group.comms[ comm_i] ) );
    }

    for ( int group_i = 0; group_i < blinkplusHelperGroupsContainer.size(); ++group_i )
    {
        for ( int comm_i = 0; comm_i < blinkplusHelperGroupsContainer.at( group_i ).num_comms; ++comm_i )
        {
            CUDACHECK(hipSetDevice( blinkplusHelperGroupsContainer.at( group_i ).devs[ comm_i ] ));
            CUDACHECK(hipFree( blinkplusHelperGroupsContainer.at( group_i ).sendbuff[ comm_i ] ));
            CUDACHECK(hipFree( blinkplusHelperGroupsContainer.at( group_i ).recvbuff[ comm_i ] ));
            NCCLCHECK( ncclCommDestroy( blinkplusHelperGroupsContainer.at( group_i ).comms[ comm_i ] ) );
        }
    }


    printf("%s:: Success \n", __func__);
    return 0;
}