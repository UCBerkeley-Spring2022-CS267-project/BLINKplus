#include <stdio.h>
#include <array>
#include <string>
#include <cstdlib>
#include <vector>
#include <cstdlib>
#include <string>
#include "hip/hip_runtime.h"
#include "nccl.h"
//#include "hip/hip_runtime_api.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

struct group_info
{
    std::vector<int> devs;
    std::vector<ncclComm_t> comms;
    std::vector<int*> sendbuff;
    std::vector<int*> recvbuff;
    std::vector<hipStream_t> streams;

    size_t num_comm;

    std::string graph_filepath;

    group_info( const char* graph_filepath_cstr, std::vector<int> devs )
    {
        if ( std::getenv(graph_filepath_cstr) == nullptr )
        {
            throw std::runtime_error("NCCL_GRAPH_FILE_CHAIN_021 not set\b");
        }
        graph_filepath = std::getenv(graph_filepath_cstr);

        this->devs = devs;
        this->num_comm = devs.size();
        this->resize( this->num_comm );
    }

    void resize( size_t num_comm )
    {
        comms.resize( num_comm );
        sendbuff.resize( num_comm );
        recvbuff.resize( num_comm );
        streams.resize( num_comm );
    }
};

void init_data( group_info& group, size_t data_size )
{
    for ( int i = 0; i < group.num_comm; ++i )
    {
        CUDACHECK(hipSetDevice( group.devs[ i ] ));
        CUDACHECK(hipMalloc( &(group.sendbuff[ i ]), data_size * sizeof(int)));
        CUDACHECK(hipMalloc( &(group.recvbuff[ i ]), data_size * sizeof(int)));
        CUDACHECK(hipMemset(  group.sendbuff[ i ], 1, data_size * sizeof(int)));
        CUDACHECK(hipMemset(  group.recvbuff[ i ], 0, data_size * sizeof(int)));
        CUDACHECK(hipStreamCreateWithFlags( &(group.streams[i]), hipStreamNonBlocking ));
    }
}

void run_broadcast( group_info& group, size_t data_size )
{
    //NCCLCHECK(ncclGroupStart());
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
        NCCLCHECK(ncclBroadcast((const void*)group.sendbuff[ i ], \
                                (void*)group.recvbuff[ i ], \
                                data_size, ncclInt, 0, \
                                group.comms[i], \
                                group.streams[i]));
    }
    //NCCLCHECK(ncclGroupEnd());
}

void run_reduce( group_info& group, size_t data_size )
{
    //NCCLCHECK(ncclGroupStart());
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
      // allreduce
      NCCLCHECK(ncclAllReduce( (const void*)group.sendbuff[ i ], \
                               (void*)group.recvbuff[ i ], \
                               data_size, ncclInt, ncclSum, \
                               group.comms[i], \
                               group.streams[i]) );
    }
    //NCCLCHECK(ncclGroupEnd());
}

void init_comm( group_info& group )
{
    setenv( "NCCL_GRAPH_FILE", group.graph_filepath.c_str() , 1 );
    NCCLCHECK(ncclCommInitAll( group.comms.data(), group.num_comm, group.devs.data() ));
}

void sync_stream( group_info& group )
{
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
      CUDACHECK(hipSetDevice( group.devs[i]));
      CUDACHECK(hipStreamSynchronize( group.streams[i]));
    }
}

void free_buffer( group_info& group )
{
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
        CUDACHECK(hipSetDevice( group.devs[i] ));
        CUDACHECK(hipFree( group.sendbuff[i] ));
        CUDACHECK(hipFree( group.recvbuff[i] ));
    }
}

void free_nccl( group_info& group )
{
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
        ncclCommDestroy( group.comms[i]);
    }
}

int main(int argc, char* argv[])
{
    printf("NCCL Version %d.%d.%d\n", NCCL_MAJOR, NCCL_MINOR, NCCL_PATCH );

    // Reference
    // https://github.com/NVIDIA/nccl/issues/574
    // https://github.com/NVIDIA/nccl/issues/217
    // https://github.com/NVIDIA/nccl/issues/195#issuecomment-473344810
    // https://github.com/NVIDIA/nccl/issues/239#issuecomment-510565429
    // https://github.com/NVIDIA/nccl/issues/315

    // set enviroment variable before run
    // this is program level setting and thus do not pollute global 
    setenv( "NCCL_PROTO", "Simple", 1);
    setenv( "NCCL_DEBUG", "Info", 1);
    setenv( "NCCL_DEBUG_SUBSYS", "ALL", 1);
    setenv( "NCCL_ALGO", "Tree", 1 ); // Tree : AllReduceTree+BroadcastRing | Ring : AllReduceRing+BroadcastRing

    // managing 4 devices
    int data_size = 64*1024*1024;

    group_info group01( "NCCL_GRAPH_FILE_CHAIN_01", std::vector<int>{0,1} );
    group_info group02( "NCCL_GRAPH_FILE_CHAIN_02", std::vector<int>{0,2} );

    // Set and initial data
    init_data( group01, data_size );
    init_data( group02, data_size );

    // Start profiling
    //hipProfilerStart();

    // Initial communicator
    printf("\n\n!!!!!Initial comm\n"); fflush(stdout);
    init_comm( group01 );
    init_comm( group02 );

    // Collective run
    printf("\n\n!!!!!Run collective\n"); fflush(stdout);
    NCCLCHECK(ncclGroupStart());
    run_broadcast( group01, data_size );
    run_reduce( group02, data_size );
    NCCLCHECK(ncclGroupEnd());

    // synchronize streams
    printf("\n\n!!!!!stream synchronize\n"); fflush(stdout);
    sync_stream( group01 );
    sync_stream( group02 );

    // End profiling
    //hipProfilerStop();

    //free device buffers
    printf("\n\n!!!!!free used buffer\n"); fflush(stdout);
    free_buffer( group01 );
    free_buffer( group02 );

    //finalizing NCCL
    printf("\n\n!!!!!free comm buffer\n"); fflush(stdout);
    free_nccl( group01 );
    free_nccl( group02 );


    printf("\n\n!!!!!Success \n");
    return 0;
}