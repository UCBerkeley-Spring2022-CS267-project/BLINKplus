#include <stdio.h>
#include <array>
#include <string>
#include <cstdlib>
#include <vector>
#include <cstdlib>
#include <string>
#include "hip/hip_runtime.h"
#include "nccl.h"

#define CUDACHECK(cmd) do {                         \
  hipError_t e = cmd;                              \
  if( e != hipSuccess ) {                          \
    printf("Failed: Cuda error %s:%d '%s'\n",       \
        __FILE__,__LINE__,hipGetErrorString(e));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)


#define NCCLCHECK(cmd) do {                         \
  ncclResult_t r = cmd;                             \
  if (r!= ncclSuccess) {                            \
    printf("Failed, NCCL error %s:%d '%s'\n",       \
        __FILE__,__LINE__,ncclGetErrorString(r));   \
    exit(EXIT_FAILURE);                             \
  }                                                 \
} while(0)

struct group_info
{
    std::vector<int> devs;
    std::vector<ncclComm_t> comms;
    std::vector<int*> sendbuff;
    std::vector<int*> recvbuff;
    std::vector<hipStream_t> streams;

    size_t num_comm;

    std::string graph_filepath;

    group_info( const char* graph_filepath_cstr, std::vector<int> devs )
    {
        if ( std::getenv(graph_filepath_cstr) == nullptr )
        {
            throw std::runtime_error("NCCL_GRAPH_FILE_CHAIN_021 not set\b");
        }
        graph_filepath = std::getenv(graph_filepath_cstr);

        this->devs = devs;
        this->num_comm = devs.size();
        this->resize( this->num_comm );
    }

    void resize( size_t num_comm )
    {
        comms.resize( num_comm );
        sendbuff.resize( num_comm );
        recvbuff.resize( num_comm );
        streams.resize( num_comm );
    }
};

void init_data( group_info& group, size_t data_size )
{
    for ( int i = 0; i < group.num_comm; ++i )
    {
        CUDACHECK(hipSetDevice( group.devs[ i ] ));
        CUDACHECK(hipMalloc( &(group.sendbuff[ i ]), data_size * sizeof(int)));
        CUDACHECK(hipMalloc( &(group.recvbuff[ i ]), data_size * sizeof(int)));
        CUDACHECK(hipMemset(  group.sendbuff[ i ], 1, data_size * sizeof(int)));
        CUDACHECK(hipMemset(  group.recvbuff[ i ], 0, data_size * sizeof(int)));
        CUDACHECK(hipStreamCreate( &(group.streams[ i ]) ));
    }
}

void init_data_helper( const group_info& group, group_info& helper_group, int helper_group_idx, int num_helper_group, size_t data_size )
{
    printf("\n");
    if ( helper_group_idx < 1 )
    {
        throw std::runtime_error("helper_group idx should start at 1\n");
    }

    for ( int i = 0; i < helper_group.num_comm; ++i )
    {
        printf("@LOG@ set helper group device %d\n", helper_group.devs[ i ] );
        CUDACHECK(hipSetDevice( helper_group.devs[ i ] ));
    
        bool use_user_buffer = false;
        for ( int j = 0; j < group.num_comm; ++j )
        {
            // Use user group buffer 
            if ( group.devs[ j ] == helper_group.devs[ i ] )
            {
                printf("@LOG@ helper group device %d use user buffer\n", helper_group.devs[ i ] );
                use_user_buffer = true;

                int offset = helper_group_idx * data_size / (num_helper_group+1);
                helper_group.sendbuff[ i ] = group.sendbuff[ i ] + offset;
                helper_group.recvbuff[ i ] = group.recvbuff[ i ] + offset;

                //CUDACHECK(hipMalloc( &(helper_group.sendbuff[ i ]), data_size / (num_helper_group+1) * sizeof(int)));
                //CUDACHECK(hipMalloc( &(helper_group.recvbuff[ i ]), data_size / (num_helper_group+1) * sizeof(int)));

                //CUDACHECK(hipMalloc( &(helper_group.recvbuff[ i ]), data_size / (num_helper_group+1) * sizeof(int)));
                //CUDACHECK(hipMemset(  helper_group.recvbuff[ i ], 0, data_size / (num_helper_group+1) * sizeof(int)));

                printf("@LOG@ user address %lld/%p, our address %lld/%p, address diff %lld, offset %d, data size %d, helper idx %d\n", \
                    group.sendbuff[ i ], group.sendbuff[ i ], helper_group.sendbuff[ i ], helper_group.sendbuff[ i ], \
                    helper_group.sendbuff[ i ] - group.sendbuff[ i ], \
                    offset, data_size, helper_group_idx );
                break;
            }
        }

        // allocate own buffer
        if ( !use_user_buffer )
        {
            printf("@LOG@ helper group device %d use internal buffer\n", helper_group.devs[ i ] );
            CUDACHECK(hipMalloc( &(helper_group.sendbuff[ i ]), data_size / (num_helper_group+1) * sizeof(int)));
            CUDACHECK(hipMalloc( &(helper_group.recvbuff[ i ]), data_size / (num_helper_group+1) * sizeof(int)));
            CUDACHECK(hipMemset(  helper_group.sendbuff[ i ], 1, data_size / (num_helper_group+1) * sizeof(int)));
            CUDACHECK(hipMemset(  helper_group.recvbuff[ i ], 0, data_size / (num_helper_group+1) * sizeof(int)));
        }

        CUDACHECK(hipStreamCreate( &(helper_group.streams[ i ]) ));
    }
}

void run_broadcast( group_info& group, size_t data_size )
{
    NCCLCHECK(ncclGroupStart());
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
        NCCLCHECK(ncclBroadcast((const void*)group.sendbuff[ i ], \
                                (void*)group.recvbuff[ i ], \
                                data_size, ncclInt, 0, \
                                group.comms[i], \
                                group.streams[i]));
    }
    NCCLCHECK(ncclGroupEnd());
}

void run_reduce( group_info& group, size_t data_size )
{
    NCCLCHECK(ncclGroupStart());
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
      // allreduce
      NCCLCHECK(ncclAllReduce( (const void*)group.sendbuff[ i ], \
                               (void*)group.recvbuff[ i ], \
                               data_size, ncclInt, ncclSum, \
                               group.comms[i], \
                               group.streams[i]) );
    }
    NCCLCHECK(ncclGroupEnd());
}

void init_comm( group_info& group )
{
    setenv( "NCCL_GRAPH_FILE", group.graph_filepath.c_str() , 1 );
    NCCLCHECK(ncclCommInitAll( group.comms.data(), group.num_comm, group.devs.data() ));
}

void sync_stream( group_info& group )
{
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
      CUDACHECK(hipSetDevice( group.devs[i] ));
      CUDACHECK(hipStreamSynchronize( group.streams[i] ));
    }
}

void free_buffer( group_info& group )
{
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
        CUDACHECK(hipSetDevice( group.devs[i] ));
        CUDACHECK(hipFree( group.sendbuff[i] ));
        CUDACHECK(hipFree( group.recvbuff[i] ));
    }
}

void free_buffer_helper( const group_info& group, group_info& helper_group )
{
    for ( int i = 0; i < helper_group.num_comm; ++i ) 
    {
        bool is_user_buffer = false;
        for ( int j = 0; j < group.num_comm; ++j )
        {
            if ( group.devs[ j ] == helper_group.devs[ i ] )
            {
                is_user_buffer = true;
                break;
            }
        }
        
        if ( !is_user_buffer )
        {
            CUDACHECK(hipSetDevice( helper_group.devs[i] ));
            CUDACHECK(hipFree( helper_group.sendbuff[i] ));
            CUDACHECK(hipFree( helper_group.recvbuff[i] ));
        }
    }
}


void free_nccl( group_info& group )
{
    for ( int i = 0; i < group.num_comm; ++i ) 
    {
        ncclCommDestroy( group.comms[i]);
    }
}

int main(int argc, char* argv[])
{
    // Reference
    // https://github.com/NVIDIA/nccl/issues/574

    // set enviroment variable before run
    // this is program level setting and thus do not pollute global 
    setenv( "NCCL_PROTO", "Simple", 1);
    //setenv( "NCCL_DEBUG", "Info", 1);
    //setenv( "NCCL_DEBUG_SUBSYS", "ALL", 1);
    setenv( "NCCL_ALGO", "Tree", 1 ); // Tree : AllReduceTree+BroadcastRing | Ring : AllReduceRing+BroadcastRing

    // managing 4 devices
    int data_size = 256*1024*1024;
    int num_helper_group = 1;

    group_info group01( "NCCL_GRAPH_FILE_CHAIN_01", std::vector<int>{0,1} );
    group_info helper_group021( "NCCL_GRAPH_FILE_CHAIN_021", std::vector<int>{0,2,1} );
    // group_info helper_group031( "NCCL_GRAPH_FILE_CHAIN_031", std::vector<int>{0,3,1} );
    // group_info helper_group0321( "NCCL_GRAPH_FILE_CHAIN_0321", std::vector<int>{0,3,2,1} );

    // Set and initial data
    init_data( group01, data_size );
    init_data_helper( group01, helper_group021, 1, num_helper_group, data_size );
    // init_data_helper( group01, helper_group031, 2, num_helper_group, data_size );
    // init_data_helper( group01, helper_group0321, 3, num_helper_group, data_size );

    // Initial communicator
    printf("\n\n!!!!!Initial comm\n"); fflush(stdout);
    init_comm( group01 );
    init_comm( helper_group021 );
    // init_comm( helper_group031 );
    // init_comm( helper_group0321 );

    // Collective run
    printf("\n\n!!!!!Run broadcast\n"); fflush(stdout);
    run_broadcast( group01, data_size / ( num_helper_group + 1 ) );
    run_broadcast( helper_group021, data_size / ( num_helper_group + 1 ));
    // run_broadcast( helper_group031, data_size / ( num_helper_group + 1 ) );
    // run_broadcast( helper_group0321, data_size / ( num_helper_group + 1 ) );

    // printf("\n\n!!!!!Run allreduce\n"); fflush( stdout );
    // run_reduce( group01, data_size / ( num_helper_group + 1 ) );
    // run_reduce( helper_group021, data_size / ( num_helper_group + 1 ) );
    // run_reduce( helper_group031, data_size / ( num_helper_group + 1 ) );
    // run_reduce( helper_group0321, data_size / ( num_helper_group + 1 ) );

    // synchronize streams
    printf("\n\n!!!!!stream synchronize user\n"); fflush(stdout);
    sync_stream( group01 );
    printf("\n\n!!!!!stream synchronize helper 1\n"); fflush(stdout);
    sync_stream( helper_group021 );
    printf("\n\n!!!!!stream synchronize helper 2\n"); fflush(stdout);
    // sync_stream( helper_group031 );
    printf("\n\n!!!!!stream synchronize helper 3\n"); fflush(stdout);
    // sync_stream( helper_group0321 );

    //free device buffers
    printf("\n\n!!!!!free used buffer\n"); fflush(stdout);
    free_buffer( group01 );
    free_buffer_helper( group01, helper_group021 );
    // free_buffer_helper( group01, helper_group031 );
    // free_buffer_helper( group01, helper_group0321 );

    //finalizing NCCL
    printf("\n\n!!!!!free comm buffer\n"); fflush(stdout);
    free_nccl( group01 );
    free_nccl( helper_group021 );
    // free_nccl( helper_group031 );
    // free_nccl( helper_group0321 );

    printf("\n\n!!!!!Success \n");
    return 0;
}